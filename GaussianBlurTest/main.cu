#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
// * Name:       STB_readWriteTest.cxx
// * Purpose:    Testing STB image libary functions

// * History:    Daire O'Neill, December 2023
// -----------------------------------------------------------------------------

#include <cmath>
#include <iostream>
#include <string>
#include <stdlib.h> 

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
//Single file image lib (header and implem in one)
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define GRAYSCALE 0
#define SEPIA 0
#define NO_FILTER 0
#define RED_FILTER 0
#define BORDER 0
#define GAUSSIAN 1

//const char* filename = "../LivingRoom.jpg";
const char* filename = "gaussVert.jpg";

int kernelRadius = 5;
//Sigma = 2
//float kernel[6] = {0.19859610213125314, 0.17571363439579307, 0.12170274650962626, 0.06598396774984912, 0.028001560233780885, 0.009300040045324049 };
//float kernel[9] = {0.1974167643837327, 0.17467018127877265, 0.12098003019548202, 0.06559213033177563, 0.027835276522173394, 0.009244812937837047, 0.002402783593203998, 0.0004886523773990668, 0.00007775057148979088};

//Sigma = 5, radius 5
float kernel[6] = {0.10852806958754817, 0.10647588402345369, 0.10055043971256167, 0.09139801540527086, 0.07996681437063455, 0.06734481169430515};
//radius 9
//float kernel[10] = {0.08318856568159615, 0.08161553140356746, 0.07707358004367856, 0.0700580949850532, 0.0612958897629078, 0.05162091532851887, 0.04184482605116835, 0.03264970400357284, 0.02452096257892869, 0.017726213001806112};

enum pxColour{
  RED = 0, 
  GREEN, 
  BLUE 
};

enum pxColour colour; 

unsigned char getPixelColour(int x, int y, int width, int height, int numChannels, pxColour colour, unsigned char *data)
{
  if (x < 0 || x >= width || y < 0 ||  y >= height || numChannels!=3 || colour < 0 || colour > 3){
    printf("Error getPixelColour: out of bounds\n");
    return 0;
  }

  else{
    return *(data + ((x + y*width) * numChannels) + (int) colour);
  }
}

void setPixelColour(int x, int y, int width, int height, int numChannels, pxColour colour, unsigned char *data, unsigned char value)
{
  if (x < 0 || x >= width || y < 0 ||  y >= height || numChannels!=3 || colour < 0 || colour > 3 ){
    printf("Error setPixelColour: out of bounds x:%d y:%d chans:%d colour%d\n", x,y,numChannels,colour);
  }

  else{
    *(data + ((x + y*width) * numChannels) + (int) colour) = value;
  }
}

int main(int argc, char **argv) {

  std::cout << "\nGaussian Blur Test\n" << std::endl;

  //Query image for info
  int inputWidth, inputHeight, inputChannels; //set by lib on load, but forced to 4 above
  int success = stbi_info(filename, &inputWidth, &inputHeight, &inputChannels);
  if(success){
    printf("Read file OK. x:%d y:%d inputChannels:%d Size:%d\n", inputWidth, inputHeight, inputChannels, (inputWidth*inputHeight*inputChannels));

  }

  else{
    printf("Error reading file\n");
  }

  if(inputChannels!=3){ //Accept 3 channels only, to simplify processing
    printf("Error: expect image with 3 input channels\n");
    return 0;
  }

  //Load image into memory. malloc done internally, returns ptr to data
  unsigned char *inputData = stbi_load(filename, &inputWidth, &inputHeight, &inputChannels, 0); //last arg is num chans to load, set 0 to load all
  int inputImageSize = (inputWidth * inputHeight * inputChannels);

  //Calculate num channels and size for output file
#if GRAYSCALE
  int outputChannels = inputChannels == 4 ? 2 : 1;
#else
  int outputChannels = inputChannels;
#endif

  int outputWidth = inputWidth;
  int outputHeight = inputHeight; //Assume we always write image of same size as input
  int outputImageSize = (outputWidth * outputHeight * outputChannels);

  printf("Allocate memory\n");
  unsigned char *outputData = new unsigned char[outputImageSize]; 
  unsigned char *tempData = new unsigned char[outputImageSize]; 
  if(outputData == NULL || tempData == NULL ) {
    printf("Unable to allocate memory for output image\n");
    exit(1);
  }

#if GRAYSCALE
  printf("Convert to gray\n"); //calc average of rgb pixels
  for(unsigned char *p = inputData, *pg = outputData; p != inputData + inputImageSize; p += inputChannels, pg += outputChannels)
  {
    *pg = (uint8_t)((*p + *(p + 1) + *(p + 2))/3.0);
  }
#elif SEPIA
  printf("Sepia filter\n");
  //Sepia filter coefficients from https://stackoverflow.com/questions/1061093/how-is-a-sepia-tone-created
  for(unsigned char *p = inputData, *pg = outputData; p != inputData + inputImageSize; p += inputChannels, pg += outputChannels)
  {
    *pg       = (uint8_t)fmin(0.393 * *p + 0.769 * *(p + 1) + 0.189 * *(p + 2), 255.0);         // writing to red pixel in target image (pg)
    *(pg + 1) = (uint8_t)fmin(0.349 * *p + 0.686 * *(p + 1) + 0.168 * *(p + 2), 255.0);         // ""         green
    *(pg + 2) = (uint8_t)fmin(0.272 * *p + 0.534 * *(p + 1) + 0.131 * *(p + 2), 255.0);         // ""         blue  
    //Note its R= (0.272 * *p), G = (0.534 * *(p + 1)), B = (0.131 * *(p + 2)). So getting each RGB component for each pixel, mult by coeff and write to output
  }
#elif NO_FILTER
  printf("No filter\n");

  unsigned char *in = inputData;
  unsigned char *out = outputData;

  for(int i = 0; i<inputImageSize; i+=inputChannels) //inputImageSize is in pixels (3 elements, RGB). But data ptr is per element (should be 3x more)
  {
    *out       = *in; 
    *(out + 1) = *(in + 1);
    *(out + 2) = *(in + 2);

    //printf("Read values: R: %d G: %d B: %d \n", *in, *(in + 1), *(in + 2));
    //printf("Wrote values: R: %d G: %d B: %d \n", *out, *(out + 1), *(out + 2));

    in += inputChannels; //move to next pixel
    out += outputChannels;
  }

#elif RED_FILTER
  printf("Red filter\n");
  float redBalance = 0.0; //range 0.0 - 255.0
  for(unsigned char *p = inputData, *pg = outputData; p != inputData + inputImageSize; p += inputChannels, pg += outputChannels)
  {
    *pg       = (uint8_t) *p * (redBalance/255.0); //Need float else any value less than 255 just goes to 0
    *(pg + 1) = *(p+1);
    *(pg + 2) = *(p+2);
  }

#elif BORDER
  printf("Draw border\n");

  unsigned char *in = inputData;
  unsigned char *out = outputData;
  int border_width = 5;
  int rowIdx = 0;
  int colIdx = 0;

  for(int i = 0; i<inputImageSize; i+=inputChannels)
  {
    /* //test - draw top half of image only
    if(i<=inputWidth*(inputHeight/2)*inputChannels){    
      *out       = *in;
      *(out + 1) = *(in + 1);
      *(out + 2) = *(in + 2);
    } */

    //Draw top and bottom borders
    if((rowIdx<border_width) || (rowIdx>(inputHeight-border_width)))
    {
      *out       = 255;
      *(out + 1) = 0;
      *(out + 2) = 0;
    }

    //Draw side borders
    else if((colIdx < border_width) || (colIdx >= (inputWidth-border_width)))
    {
      *out       = 255;
      *(out + 1) = 0;
      *(out + 2) = 0;

      //printf("colIdx: %d\n", colIdx);
    }

    //Write image
    else
    {
      *out       = *in;
      *(out + 1) = *(in + 1);
      *(out + 2) = *(in + 2);
    }

    colIdx++;  //already counting cols each loop iteration (1 col = 1 px = 3 chans, i+=inputChans)

    if(i%(inputWidth*inputChannels) == 0) //Count rows
    {
      //printf("rowIdx: %d\n", rowIdx);
      rowIdx++;
      colIdx = 0; //new row, reset cols
    }

    in += inputChannels; //move to next pixel
    out += outputChannels;
  }

  //Target test
  for (int val=0;val<100;val++)
  {
    setPixelColour(270+val,240,inputWidth,inputHeight,inputChannels,RED,outputData,255); //horiz: 320-50 = 270
    setPixelColour(320,190+val,inputWidth,inputHeight,inputChannels,GREEN,outputData,255); //vert:  240-50 = 190
  }

#elif GAUSSIAN
  unsigned char output_red_h = 0, output_green_h = 0, output_blue_h = 0;
  unsigned char output_red_v = 0, output_green_v = 0, output_blue_v = 0;

  unsigned char *in = inputData;
  int border_width = kernelRadius+1;
  int rowIdx = 0;
  int colIdx = 0;


  while(rowIdx<inputHeight) //Scan rows top to bottom
  {
    if((rowIdx < border_width) || (rowIdx>(inputHeight-border_width))) //top and bottom borders
    {
      setPixelColour(colIdx, rowIdx, inputWidth, inputHeight, inputChannels, RED, tempData, 255);
    }

    else if((colIdx < border_width) || (colIdx >= (inputWidth-border_width))) //side borders
    {
      setPixelColour(colIdx, rowIdx, inputWidth, inputHeight, inputChannels, RED, tempData, 255);
    }

    else //blur
    {
      //Get middle (current) pixel colour
      output_red_h = getPixelColour(colIdx, rowIdx, inputWidth, inputHeight, inputChannels, RED, in) * kernel[0];
      output_green_h = getPixelColour(colIdx, rowIdx, inputWidth, inputHeight, inputChannels, GREEN, in) * kernel[0];
      output_blue_h = getPixelColour(colIdx, rowIdx, inputWidth, inputHeight, inputChannels, BLUE, in) * kernel[0];
  
      //Add horizontal px
      for(int k = 1; k<=kernelRadius; k++)
      {
        //printf("k=%d kernel[k]=%f\n", k, kernel[k]);
        output_red_h += getPixelColour(colIdx+k, rowIdx, inputWidth, inputHeight, inputChannels, RED, in) * kernel[k]; //px to the right 
        output_red_h += getPixelColour(colIdx-k, rowIdx, inputWidth, inputHeight, inputChannels, RED, in) * kernel[k]; //px to the left

        output_green_h += getPixelColour(colIdx+k, rowIdx, inputWidth, inputHeight, inputChannels, GREEN, in) * kernel[k];
        output_green_h += getPixelColour(colIdx-k, rowIdx, inputWidth, inputHeight, inputChannels, GREEN, in) * kernel[k];

        output_blue_h += getPixelColour(colIdx+k, rowIdx, inputWidth, inputHeight, inputChannels, BLUE, in) * kernel[k];
        output_blue_h += getPixelColour(colIdx-k, rowIdx, inputWidth, inputHeight, inputChannels, BLUE, in) * kernel[k];
      }

      //Write horiz pixels
      setPixelColour(colIdx, rowIdx, inputWidth, inputHeight, inputChannels, RED, tempData, (output_red_h));
      setPixelColour(colIdx, rowIdx, inputWidth, inputHeight, inputChannels, GREEN, tempData, (output_green_h));
      setPixelColour(colIdx, rowIdx, inputWidth, inputHeight, inputChannels, BLUE, tempData, (output_blue_h));
    }

    colIdx++; //Move to next pixel in row
    if(colIdx>=inputWidth) 
    {
      rowIdx++;
      colIdx = 0; //new row, reset cols
    }
  }
   
    rowIdx = 0;
    colIdx = 0;

    //Use horiz blur data as input for vert blur
    while(rowIdx<inputHeight) //Scan rows top to bottom
    {
      if((rowIdx < border_width) || (rowIdx>(inputHeight-border_width))) //top and bottom borders
      {
        setPixelColour(colIdx, rowIdx, inputWidth, inputHeight, inputChannels, RED, outputData, 255);
      }

      else if((colIdx < border_width) || (colIdx >= (inputWidth-border_width))) //side borders
      {
        setPixelColour(colIdx, rowIdx, inputWidth, inputHeight, inputChannels, RED, outputData, 255);
      }

      else //blur
      {
        output_red_v = getPixelColour(colIdx, rowIdx, inputWidth, inputHeight, inputChannels, RED, tempData) * kernel[0];
        output_green_v = getPixelColour(colIdx, rowIdx, inputWidth, inputHeight, inputChannels, GREEN, tempData) * kernel[0];
        output_blue_v = getPixelColour(colIdx, rowIdx, inputWidth, inputHeight, inputChannels, BLUE, tempData) * kernel[0];

        for(int k = 1; k<=kernelRadius;k++)
        {
          output_red_v += getPixelColour(colIdx, rowIdx+k, inputWidth, inputHeight, inputChannels, RED, tempData) * kernel[k]; //px above
          output_red_v += getPixelColour(colIdx, rowIdx-k, inputWidth, inputHeight, inputChannels, RED, tempData) * kernel[k]; //px below

          output_green_v += getPixelColour(colIdx, rowIdx+k, inputWidth, inputHeight, inputChannels, GREEN, tempData) * kernel[k];
          output_green_v += getPixelColour(colIdx, rowIdx-k, inputWidth, inputHeight, inputChannels, GREEN, tempData) * kernel[k];

          output_blue_v += getPixelColour(colIdx, rowIdx+k, inputWidth, inputHeight, inputChannels, BLUE, tempData) * kernel[k];
          output_blue_v += getPixelColour(colIdx, rowIdx-k, inputWidth, inputHeight, inputChannels, BLUE, tempData) * kernel[k];
        }

        //Write vert pixels
        setPixelColour(colIdx, rowIdx, inputWidth, inputHeight, inputChannels, RED, outputData, (output_red_v));
        setPixelColour(colIdx, rowIdx, inputWidth, inputHeight, inputChannels, GREEN, outputData, (output_green_v));
        setPixelColour(colIdx, rowIdx, inputWidth, inputHeight, inputChannels, BLUE, outputData, (output_blue_v));
      }

      colIdx++; //Move to next pixel in row
      if(colIdx>=inputWidth)
      {
        rowIdx++;
        colIdx = 0; //new row, reset cols
      }
    }
#endif

  //Write result to file
  printf("Write to file\n");
  success = stbi_write_jpg("gauss5.jpg", outputWidth, outputHeight, outputChannels, outputData, 100); //last arg is quality, 1-100
  if(success){
    printf("Wrote file OK! x:%d y:%d channels:%d\n", inputWidth, inputHeight, outputChannels);
  }

  else{
    printf("Error writing file\n");
  }

  stbi_image_free(inputData);
  delete(outputData);
  delete(tempData);

  return (EXIT_SUCCESS);
}
