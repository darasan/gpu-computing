#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
// * Name:       STB_readWriteTest.cxx
// * Purpose:    Testing STB image libary functions

// * History:    Daire O'Neill, December 2023
// -----------------------------------------------------------------------------

#include <cmath>
#include <iostream>
#include <string>
#include <stdlib.h> 

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
//Single file image lib (header and implem in one)
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define GRAYSCALE 0
#define SEPIA 0
#define NO_FILTER 0
#define RED_FILTER 1
#define GAUSSIAN 0

const char* filename = "../LivingRoom.jpg";

int main(int argc, char **argv) {

  std::cout << "\nGaussian Blur Test\n" << std::endl;

  //Query image for info
  int inputWidth, inputHeight, inputChannels; //set by lib on load, but forced to 4 above
  int success = stbi_info(filename, &inputWidth, &inputHeight, &inputChannels);
  if(success){
    printf("Read file OK. x:%d y:%d inputChannels:%d Size:%d\n", inputWidth, inputHeight, inputChannels, (inputWidth*inputHeight*inputChannels));

  }
  else{
    printf("Error reading file\n");
  }

  //Load image into memory. malloc done internally, returns ptr to data
  unsigned char *inputData = stbi_load(filename, &inputWidth, &inputHeight, &inputChannels, 0); //last arg is num chans to load, set 0 to load all
  int inputImageSize = (inputWidth * inputHeight * inputChannels);

  //Calculate num channels and size for output file
#if GRAYSCALE
  int outputChannels = inputChannels == 4 ? 2 : 1;
#else
  int outputChannels = inputChannels;
#endif

  int outputWidth = inputWidth;
  int outputHeight = inputHeight; //Assume we always write image of same size as input
  int outputImageSize = (outputWidth * outputHeight * outputChannels);

  printf("Allocate memory\n");
  unsigned char *outputData = new unsigned char[outputImageSize]; 
  if(outputData == NULL) {
    printf("Unable to allocate memory for output image\n");
    exit(1);
  }

#if GRAYSCALE
  printf("Convert to gray\n"); //calc average of rgb pixels
  for(unsigned char *p = inputData, *pg = outputData; p != inputData + inputImageSize; p += inputChannels, pg += outputChannels)
  {
    *pg = (uint8_t)((*p + *(p + 1) + *(p + 2))/3.0);
    if(inputChannels == 4) {
      *(pg + 1) = *(p + 3);
    }
  }
#elif SEPIA
  printf("Sepia filter\n");
  //Sepia filter coefficients from https://stackoverflow.com/questions/1061093/how-is-a-sepia-tone-created
  for(unsigned char *p = inputData, *pg = outputData; p != inputData + inputImageSize; p += inputChannels, pg += outputChannels)
  {
    *pg       = (uint8_t)fmin(0.393 * *p + 0.769 * *(p + 1) + 0.189 * *(p + 2), 255.0);         // writing to red pixel in target image (pg)
    *(pg + 1) = (uint8_t)fmin(0.349 * *p + 0.686 * *(p + 1) + 0.168 * *(p + 2), 255.0);         // ""         green
    *(pg + 2) = (uint8_t)fmin(0.272 * *p + 0.534 * *(p + 1) + 0.131 * *(p + 2), 255.0);         // ""         blue  
    //Not very well written, note its R= (0.272 * *p), G = (0.534 * *(p + 1)), B = (0.131 * *(p + 2)). So getting each RGB component for each pixel, mult by coeff and write to output
    if(inputChannels == 4) {
      //Just copy alpha channel as is
      *(pg + 3) = *(p + 3);
    }
  }
#elif NO_FILTER
  printf("No filter\n");
  for(unsigned char *p = inputData, *pg = outputData; p != inputData + inputImageSize; p += inputChannels, pg += outputChannels)
  {
    *pg       = *p;
    *(pg + 1) = *(p+1);
    *(pg + 2) = *(p+2);
    if(inputChannels == 4) {
      //Just copy alpha channel as is
      *(pg + 3) = *(p + 3);
    }
    //printf("Orig values: R: %d G: %d B: %d \n", *pg, *(pg + 1), *(pg + 2));
  }

#elif RED_FILTER
  printf("Red filter\n");
  float redBalance = 0.0; //range 0.0 - 255.0
  for(unsigned char *p = inputData, *pg = outputData; p != inputData + inputImageSize; p += inputChannels, pg += outputChannels)
  {
    *pg       = (uint8_t) *p * (redBalance/255.0); //Need float else any value less than 255 just goes to 0
    *(pg + 1) = *(p+1);
    *(pg + 2) = *(p+2);
    if(inputChannels == 4) {
      //Just copy alpha channel as is
      *(pg + 3) = *(p + 3);
    }
  }

#endif

  //Write result to file
  printf("Write to file\n");
  success = stbi_write_jpg("outputFile.jpg", outputWidth, outputHeight, outputChannels, outputData, 80); //last arg is quality, 1-100
  if(success){
    printf("Wrote file OK! x:%d y:%d channels:%d\n", inputWidth, inputHeight, outputChannels);
  }

  else{
    printf("Error writing file\n");
  }

  stbi_image_free(inputData);
  delete(outputData);

  return (EXIT_SUCCESS);
}
